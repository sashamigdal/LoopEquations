﻿#include <cassert>
#include <utility>
#include <random>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
 // for Intellisense
#include <hip/hip_complex.h>

using namespace std::string_literals;

// no noticeable difference
#define USE_DOUBLE 1

#if USE_DOUBLE
using cuAnyComplex = hipDoubleComplex;
using real = double;
#   define make_hipComplex make_hipDoubleComplex
#   define cuCrealr hipCreal
#   define cuCimagr hipCimag
#   define cuCsubr hipCsub
#   define cuCabsr hipCabs
#   define curand_uniform_real hiprand_uniform_double
#else
using cuAnyComplex = cuFloatComplex;
using real = float;
#   define make_cuComplex make_cuFloatComplex
#   define cuCrealr cuCrealf
#   define cuCimagr cuCimagf
#   define cuCsubr cuCsubf
#   define cuCabsr cuCabsf
#   define curand_uniform_real curand_uniform
#endif

void CheckErrorCode( hipError_t err_code ) {
    if ( err_code ) {
        std::cerr << "CUDA ERROR " << err_code << " happened" << std::endl;
    }
}

__device__ static inline cuAnyComplex expi( real a ) {
    real sin_a, cos_a;
    sincos(a, &sin_a, &cos_a);
    return make_hipComplex( cos_a, sin_a );
}

__device__ static inline cuAnyComplex& operator+= ( cuAnyComplex& z, cuAnyComplex w ) {
    z.x += w.x;
    z.y += w.y;
    return z;
}

__device__ static inline cuAnyComplex operator- ( cuAnyComplex z, cuAnyComplex w ) {
    return cuCsubr( z, w );
}

__device__ static inline cuAnyComplex operator/ ( cuAnyComplex z, real a ) {
    return make_hipComplex( cuCrealr(z) / a, cuCimagr(z) / a );
}

__device__ static inline cuAnyComplex& operator/= ( cuAnyComplex& z, real a ) {
    z.x /= a;
    z.y /= a;
    return z;
}

class cudaRandomWalker {
    friend __global__ void DoWorkKernel( cudaRandomWalker* walkers, real* S_mn, int device_id );
public:
    __device__ void Init( std::int64_t N_pos, std::int64_t N_neg ) {
        this->N_pos = N_pos;
        this->N_neg = N_neg;
        alpha = 0;
        hiprand_init( blockIdx.x * blockDim.x + threadIdx.x, threadIdx.x, 0, &gen );
    }

    __device__ int Advance() {
        int sigma = RandomSign();
        (sigma == 1 ? N_pos : N_neg)--;
        alpha += sigma;
        return sigma;
    }

    __device__ std::int64_t get_alpha() const { return alpha; }

private:
    __device__ int RandomSign() {
        return (curand_uniform_real(&gen) * real(N_pos + N_neg) <= N_neg) ? -1 : 1;
    }

    std::int64_t N_pos;
    std::int64_t N_neg;
    std::int64_t alpha; // alpha[i]
    hiprandState_t gen;
};

static __global__ void DoWorkKernel( cudaRandomWalker* walkers, std::int64_t* ns, std::int64_t* ms,
                                     std::int64_t* N_poss, std::int64_t* N_negs, real* betas, /*OUT*/ real* Ss, /*OUT*/ real* o_os )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //int tid = ((blockI blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    cudaRandomWalker& walker = walkers[tid];
    std::int64_t n = ns[tid];
    std::int64_t m = ms[tid];
    std::int64_t N_pos = N_poss[tid];
    std::int64_t N_neg = N_negs[tid];
    real beta = betas[tid];
    real& S = Ss[tid];
    real& o_o = o_os[tid];

    assert(n < m);
    std::int64_t M = N_pos + N_neg;
    int sigma_n, sigma_m, alpha_m, alpha_n;
    cuAnyComplex S_nm = make_hipComplex(0, 0);
    cuAnyComplex S_mn = make_hipComplex(0, 0);

    walker.Init( N_pos, N_neg );
    std::int64_t i = 0;
    for ( ; i != n; i++ ) { // i = [0; n)
        S_mn += expi( walker.get_alpha() * beta );
        walker.Advance();
    }

    alpha_n = walker.get_alpha();
    S_nm += expi( alpha_n * beta );
    sigma_n = walker.Advance(); // i = n
    for ( i++; i != m; i++ ) { // i = (n, m)
        S_nm += expi( walker.get_alpha() * beta );
        walker.Advance();
    }

    alpha_m = walker.get_alpha();
    S_mn += expi(alpha_m * beta);
    sigma_m = walker.Advance(); // i = m
    for ( i++; i != M; i++ ) { // i = (m, M)
        S_mn += expi(walker.get_alpha() * beta);
        walker.Advance();
    }

    /*
    -\frac{1}{2} \cot ^2\left(\frac{\beta }{2}\right) \sigma _m \sigma _n \sin ^2\left(\frac{1}{4} \left(2 \alpha _m+\beta  \left(\sigma _m-\sigma _n\right)-2 \alpha _n\right)\right)
    */
    o_o = -M * (M - 1) / 2 * sigma_n * sigma_m / (2 * pow(tan(beta / 2), real(2.0))) * pow( sin(beta / 4 * (2 * (alpha_m - alpha_n) + sigma_m - sigma_n)), real(2.0) );

    S_nm /= real(m - n);
    S_mn /= real(n + M - m);
    S = cuCabsr((S_nm - S_mn) / (2 * sin(beta / 2)));
}

template <class T>
struct pair_ptr {
    T* host_ptr;
    T* device_ptr;

    void allocate( size_t size ) {
        host_ptr = new T[size];
        CheckErrorCode( hipMalloc( (void**)&device_ptr, size * sizeof(T) ) );
    }

    ~pair_ptr() {
        CheckErrorCode( hipFree(device_ptr) );
        delete[] host_ptr;
    }

    void CopyToDevice( size_t size ) {
        CheckErrorCode( hipMemcpy( device_ptr, host_ptr, size * sizeof(T), hipMemcpyHostToDevice ) );
    }

    void CopyFromDevice( size_t size ) {
        CheckErrorCode( hipMemcpy( host_ptr, device_ptr, size * sizeof(T), hipMemcpyDeviceToHost ) );
    }
};

inline void gpuAssert( hipError_t code, const char *file, int line, bool abort=false ) {
    if ( code != hipSuccess ) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#define chkWrap(fn) \
template <typename... T>\
__host__ void chk##fn( T&&... args ) {\
    gpuErrchk( fn( std::forward<T>(args)... ) );\
}

chkWrap(hipEventCreate)
chkWrap(hipEventDestroy)
chkWrap(hipEventElapsedTime)
chkWrap(hipEventRecord)
chkWrap(hipGetDeviceProperties)

double benchmark( int gridSize, int nThreads, int device ) {
    const int totThreads = gridSize * nThreads;
    CheckErrorCode( hipSetDevice(device) );

    pair_ptr<cudaRandomWalker> walkers;
    pair_ptr<std::int64_t> ns, ms, N_poss, N_negs;
    pair_ptr<real> betas, Ss, o_os;

    walkers.allocate(totThreads);
    ns.allocate(totThreads);
    ms.allocate(totThreads);
    N_poss.allocate(totThreads);
    N_negs.allocate(totThreads);
    betas.allocate(totThreads);
    Ss.allocate(totThreads);
    o_os.allocate(totThreads);

    std::mt19937_64 gen;
    std::int64_t M = 1 << 15;
    hipDeviceProp_t devProp;
    chkcudaGetDeviceProperties( &devProp, device );
    const int warpSize = devProp.warpSize;
    if ( totThreads < warpSize ) { return -1; }
    for ( size_t i = 0; i != totThreads / warpSize; i++ ) {
        std::uniform_int_distribution<std::int64_t> unif_M( 1, M );
        std::uniform_int_distribution<std::int64_t> unif_M1( 1, M - 1 );
        std::int64_t n = unif_M(gen) - 1;
        std::int64_t m = (n + unif_M1(gen)) % M;
        if ( n > m ) {
            std::swap( n, m );
        }
        for ( int j = 0; j != warpSize; j++ ) {
            size_t idx = i * warpSize + j;
            ns.host_ptr[idx] = n;
            ms.host_ptr[idx] = m;
            N_poss.host_ptr[idx] = M / 2;
            N_negs.host_ptr[idx] = M / 2;
            betas.host_ptr[idx] = real(0.1);
        }
    }

    walkers.CopyToDevice(totThreads);
    ns.CopyToDevice(totThreads);
    ms.CopyToDevice(totThreads);
    N_poss.CopyToDevice(totThreads);
    N_negs.CopyToDevice(totThreads);
    betas.CopyToDevice(totThreads);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    chkcudaEventCreate( &start );
    chkcudaEventCreate( &stop );
    chkcudaEventRecord( start, hipStream_t(0) );

    DoWorkKernel<<<gridSize, nThreads>>>( walkers.device_ptr, ns.device_ptr, ms.device_ptr, N_poss.device_ptr, N_negs.device_ptr,
                                         betas.device_ptr, Ss.device_ptr, o_os.device_ptr );

    if ( hipGetLastError() != hipSuccess ) { return -1; }
    if ( hipDeviceSynchronize() != hipSuccess ) { return -1; }
    chkcudaEventRecord( stop, hipStream_t(0) );
    if ( hipEventSynchronize(stop) != hipSuccess ) { return -1; }

    chkcudaEventElapsedTime( &gpuTime, start, stop );

    double speed = totThreads * M / gpuTime * 1e3;

    chkcudaEventDestroy(start);
    chkcudaEventDestroy(stop);

    Ss.CopyFromDevice(totThreads);
    o_os.CopyFromDevice(totThreads);

    //for ( size_t i = 0; i != std::min(30, totThreads); i++ ) {
    //    std::cout << Ss.host_ptr[i] << '\t' << o_os.host_ptr[i] << std::endl;
    //}
    return speed;
}

int main() {
    int device = 0;
    int best_gridSize = 1;
    int best_nThreads = 1;
    double best_speed = 1;
    std::cout << "gridSize" << '\t' << "nThreads" << '\t' << "speed" << std::endl;
    hipDeviceProp_t devProp;
    chkcudaGetDeviceProperties( &devProp, device );
    std::cout << "Running on GPU \"" << devProp.name << "\"" << std::endl;
    if ( devProp.name == "NVIDIA GeForce GTX 1080 Ti"s ) {
        device = 1;
    }
    const int warpSize = devProp.warpSize;

    for ( int gridSize = 1; gridSize <= 1 << 12; gridSize *= 2 )
    //int gridSize = 4096;
    {
        for ( int nThreads = 1; nThreads <= 512; nThreads *= 2 )
        //int nThreads = 128;
        {
            if ( gridSize * nThreads < warpSize ) { continue; }
            double speed = benchmark( gridSize, nThreads, device );
            if ( speed == -1 ) {
                std::cout << "Fail for " << gridSize << '\t' << nThreads << std::endl;
            } else {
                std::cout << gridSize << '\t' << nThreads << '\t' << speed;
                if ( speed > best_speed ) {
                    best_speed = speed;
                    best_gridSize = gridSize;
                    best_nThreads = nThreads;
                    std::cout << "\t(new best)";
                }
                std::cout << std::endl;
            }
        }
    }
    std::cout << "--------------------------------------------\n";
    std::cout << "The best is " << best_gridSize << "x" << best_nThreads << " giving " << best_speed << " speed\n";
    return 0;
}
