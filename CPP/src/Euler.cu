
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 100
__global__
void add(int*a,int*b){
int i=blockIdx.x;
if(i<N){b[i]=2*a[i];}
}
int main(){
int ha[N],hb[N];
int *da,*db;
hipMalloc((void**)&da,N*sizeof(int));
hipMalloc((void**)&db,N*sizeof(int));
for(int i=0;i<N;i++){ha[i]=i;}
hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);
    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }//
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);
    return 0;
}